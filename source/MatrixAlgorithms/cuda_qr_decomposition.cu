#include "hip/hip_runtime.h"
extern "C" {
#include "cuda_qr_decomposition.h"
}

#pragma region SingleCore

__global__ void cuda_matrix_qr_decomposition_single_core_kernel(
    device_matrix_t matrix, float *diagonal, float *c, int dimension,
    bool *is_singular) {
    float column_length;  // sigma in book
    float column_length_squared, element;
    int n = dimension;
    float scale;
    *is_singular = false;

    // for every column
    for (int k = 0; k < n; k++) {
        scale = 0.0f;
        // scale is the max absolute value of the column
        for (int i = k; i < n; i++) {
            scale = fmaxf(scale, fabsf(matrix[INDEX(i, k, n)]));
        }
        if (scale == 0.0) {
            *is_singular = true;
            c[k] = diagonal[k] = 0.0f;
            continue;
        }
        // Normalize column
        for (int i = k; i < n; i++) matrix[INDEX(i, k, n)] /= scale;

        // column length below diagonal
        column_length_squared = 0.0f;  // sum in book.
        for (int i = k; i < n; i++) {
            element = matrix[INDEX(i, k, n)];
            column_length_squared += element * element;
        }

        // column length below diagonal, with the sign of diagonal k
        column_length =
            SIGN(sqrtf(column_length_squared), matrix[INDEX(k, k, n)]);

        // add column length to diagonal k
        matrix[INDEX(k, k, n)] += column_length;

        c[k] = matrix[INDEX(k, k, n)] * column_length;

        diagonal[k] = -scale * column_length;

        // Calculate Q[k] = I - (u[k] (x) u[k]) / c[k]
        for (int j = k + 1; j < n; j++) {
            // inner product for column j below diagonal
            float inner_product = 0.0f;
            for (int i = k; i < n; i++) {
                inner_product +=
                    matrix[(INDEX(i, k, n))] * matrix[(INDEX(i, j, n))];
            }

            // division
            float tau = inner_product / c[k];

            for (int i = k; i < n; i++) {
                matrix[(INDEX(i, j, n))] -= tau * matrix[(INDEX(i, k, n))];
            }
        }
    }

    if (!*is_singular) *is_singular = diagonal[n - 1] == 0.0f;
}

bool cuda_qr_decomposition_runner(matrix_t *matrix, float *diagonal, float *c,
    void (*kernel)(device_matrix_t, float *, float *, int, bool *),
    dim3 grid_size, dim3 block_size) {
    device_matrix_t device_matrix =
        cuda_matrix_init(matrix->rows, matrix->columns);
    cuda_matrix_host_to_device(device_matrix, matrix);

    size_t diagonal_size = sizeof(float) * matrix->columns;

    float *device_diagonal;
    hipMalloc(&device_diagonal, diagonal_size);

    float *device_c;
    hipMalloc(&device_c, diagonal_size);

    bool *device_is_singular;
    hipMalloc(&device_is_singular, sizeof(bool));

    kernel<<<grid_size, block_size>>>(device_matrix, device_diagonal, device_c,
        matrix->columns, device_is_singular);

    bool is_singular;
    hipMemcpy(
        &is_singular, device_is_singular, sizeof(bool), hipMemcpyDeviceToHost);
    cuda_matrix_device_to_host(matrix, device_matrix);
    hipMemcpy(
        diagonal, device_diagonal, diagonal_size, hipMemcpyDeviceToHost);
    hipMemcpy(c, device_c, diagonal_size, hipMemcpyDeviceToHost);

    cuda_matrix_free(device_matrix);
    hipFree(device_diagonal);
    hipFree(device_c);
    hipFree(device_is_singular);

    return is_singular;
}

bool cuda_matrix_qr_decomposition_single_core_adapter(
    algorithm_arg_t *matrix, algorithm_arg_t *diagonal, algorithm_arg_t *c) {
    return cuda_matrix_qr_decomposition_single_core(
        matrix->matrix, diagonal->vector, c->vector);
}

bool cuda_matrix_qr_decomposition_single_core(
    matrix_t *matrix, float *diagonal, float *c) {
    return cuda_qr_decomposition_runner(matrix, diagonal, c,
        &(cuda_matrix_qr_decomposition_single_core_kernel), dim3(1), dim3(1));
}

#pragma endregion

#pragma region MultiCoreManyKernalLaunches

bool cuda_matrix_qr_decomposition_parallel_max_adapter(
    algorithm_arg_t *matrix, algorithm_arg_t *diagonal, algorithm_arg_t *c) {
    return cuda_matrix_qr_decomposition_parallel_max(
        matrix->matrix, diagonal->vector, c->vector);
}

#define ELEMENTS_PR_THREAD 4
#define BLOCK_SIZE 4

__device__ float cuda_max_absolute(float a, float b) {
    return fmaxf(fabsf(a), fabsf(b));
}

__device__ float cuda_add(float a, float b) { return a + b; }

__device__ void cuda_max_value(
    float *scale, const float *values, int grid_size) {
    float max = values[0];
    for (int i = 1; i < grid_size; i++) {
        if (values[i] > max) max = values[i];
    }
    *scale = max;
}

__device__ void cuda_check_singularity(
    float scale, bool *is_singular, float *c, float *diagonal, int k) {
    if (scale == 0.0f) {
        *is_singular = true;
        c[k] = diagonal[k] = 0.0f;
    }
}

__global__ void cuda_get_max_value_and_check_singularity_kernel(float *scale,
    const float *values, int grid_size, bool *is_singular, float *c,
    float *diagonal, int k) {
    cuda_max_value(scale, values, grid_size);
    cuda_check_singularity(*scale, is_singular, c, diagonal, k);
}

__device__ void cuda_sum(float *sum, const float *values, int grid_size) {
    float accumulator = 0;
    for (int i = 0; i < grid_size; i++) accumulator += values[i];
    *sum = accumulator;
}

__device__ void cuda_parallel_reduction(
    float *cache, int cache_index, reducer_t reduce) {
    int split_index = blockDim.x;
    while (split_index != 0) {
        split_index /= 2;
        if (cache_index < split_index)
            cache[cache_index] =
                reduce(cache[cache_index], cache[cache_index + split_index]);

        __syncthreads();
    }
}

__device__ int get_index(int starting_index, int dimension) {
    int thread_start = threadIdx.x * dimension;
    int block_start = blockIdx.x * BLOCK_SIZE * ELEMENTS_PR_THREAD * dimension;
    return starting_index + thread_start + block_start;
}

__global__ void cuda_parallel_max_kernel(float *blocks, device_matrix_t matrix,
    int element_count, int k, int starting_index, int dimension) {
    __shared__ float cache[BLOCK_SIZE];
    int i = get_index(starting_index, dimension);
    int increment = dimension * BLOCK_SIZE;
    int cache_index = threadIdx.x;
    float thread_max = fabsf(matrix[starting_index]);

    for (int e = 0; e < ELEMENTS_PR_THREAD; e++) {
        if (i >= element_count) break;
        thread_max = cuda_max_absolute(thread_max, matrix[i]);
        i += increment;
    }

    cache[cache_index] = thread_max;
    __syncthreads();
    cuda_parallel_reduction(cache, cache_index, cuda_max_absolute);
    if (cache_index == 0) blocks[blockIdx.x] = cache[0];
}

__global__ void cuda_parallel_sum_of_products_kernel(float *blocks,
    device_matrix_t matrix, int element_count, int starting_index_1,
    int starting_index_2, int dimension) {
    __shared__ float cache[BLOCK_SIZE];
    int i = get_index(starting_index_1, dimension);
    int j = get_index(starting_index_2, dimension);
    int increment = dimension * BLOCK_SIZE;
    int cache_index = threadIdx.x;
    float sum = 0;

    for (int e = 0; e < ELEMENTS_PR_THREAD; e++) {
        if (i >= element_count) break;

        sum = cuda_add(sum, matrix[i] * matrix[j]);
        i += increment;
        j += increment;
    }

    cache[cache_index] = sum;
    __syncthreads();
    cuda_parallel_reduction(cache, cache_index, cuda_add);
    if (cache_index == 0) blocks[blockIdx.x] = cache[0];
}

__global__ void initialize_singularity(bool *is_singular) {
    *is_singular = false;
}

__global__ void cuda_scale_column(device_matrix_t matrix, float *device_scale,
    int k, int starting_index, int dimension, int element_count) {
    float scale = *device_scale;
    int i = get_index(starting_index, dimension);
    int increment = dimension * BLOCK_SIZE;

    for (int e = 0; e < ELEMENTS_PR_THREAD; e++) {
        if (i >= element_count) break;
        matrix[i] /= scale;
        i += increment;
    }
}

__global__ void cuda_matrix_qr_decomposition_kernel(float *blocks, int grid_size, device_matrix_t matrix,
    float *diagonal, float *c, int dimension, int k, float *scale_in_memory,
    float *squared_column_length) {

    cuda_sum(squared_column_length, blocks, grid_size);
    
    int diagonal_index = INDEX(k, k, dimension);
    float column_length =
        SIGN(sqrtf(*squared_column_length), matrix[diagonal_index]);

    matrix[diagonal_index] += column_length;
    c[k] = matrix[diagonal_index] * column_length;
    diagonal[k] = -*scale_in_memory * column_length;
}

__device__ void cuda_compute_tau(
    float *tau, float *inner_product, float *c, int k) {
    *tau = *inner_product / c[k];
}

__global__ void cuda_find_inner_product_and_compute_tau(
    float *inner_product, float *blocks, int grid_size,
    float *tau, float *c, int k) {
    cuda_sum(inner_product, blocks, grid_size);
    cuda_compute_tau(tau, inner_product, c, k);
}

__global__ void cuda_subtract_tau_product(device_matrix_t matrix,
    float *device_tau, float *c, int k, int j, int starting_index,
    int dimension, int element_count) {
    float tau = *device_tau;
    int i = get_index(starting_index, dimension);
    int offset = j - k;
    int increment = dimension * BLOCK_SIZE;

    for (int e = 0; e < ELEMENTS_PR_THREAD; e++) {
        if (i >= element_count) break;
        matrix[i + offset] -= tau * matrix[i];
        i += increment;
    }
}

bool cuda_matrix_qr_decomposition_parallel_max(
    matrix_t *matrix, float *diagonal, float *c) {
    device_matrix_t device_matrix =
        cuda_matrix_init(matrix->rows, matrix->columns);
    cuda_matrix_host_to_device(device_matrix, matrix);
    size_t diagonal_size = sizeof(float) * matrix->columns;

    float *device_diagonal;
    hipMalloc(&device_diagonal, diagonal_size);

    float *device_c;
    hipMalloc(&device_c, diagonal_size);

    bool *device_is_singular;
    hipMalloc(&device_is_singular, sizeof(bool));
    initialize_singularity<<<1, 1>>>(device_is_singular);

    float *device_scale;
    hipMalloc(&device_scale, sizeof(float));

    float *device_squared_column_length;
    hipMalloc(&device_squared_column_length, sizeof(float));

    float *device_inner_product;
    hipMalloc(&device_inner_product, sizeof(float));

    float *device_tau;
    hipMalloc(&device_tau, sizeof(float));

    int dimension = matrix->columns;
    int element_count = matrix->columns * matrix->rows;
    int grid_size = (dimension + ELEMENTS_PR_THREAD * BLOCK_SIZE - 1) /
                    (ELEMENTS_PR_THREAD * BLOCK_SIZE);

    float *device_blocks;
    hipMalloc(&device_blocks, sizeof(float) * grid_size);

    int starting_index;

    for (int k = 0; k < dimension; k++) {
        grid_size = (dimension - k + ELEMENTS_PR_THREAD * BLOCK_SIZE - 1) /
                    (ELEMENTS_PR_THREAD * BLOCK_SIZE);

        starting_index = INDEX(k, k, dimension);

        cuda_parallel_max_kernel<<<grid_size, BLOCK_SIZE>>>(device_blocks,
            device_matrix, element_count, k, starting_index, dimension);
        hipDeviceSynchronize();

        cuda_get_max_value_and_check_singularity_kernel<<<1, 1>>>(device_scale,
            device_blocks, grid_size, device_is_singular, device_c,
            device_diagonal, k);
        // hipDeviceSynchronize();

        cuda_scale_column<<<grid_size, BLOCK_SIZE>>>(device_matrix,
            device_scale, k, starting_index, dimension, element_count);
        hipDeviceSynchronize();

        cuda_parallel_sum_of_products_kernel<<<grid_size, BLOCK_SIZE>>>(
            device_blocks, device_matrix, element_count, starting_index,
            starting_index, dimension);
        hipDeviceSynchronize();

        cuda_matrix_qr_decomposition_kernel<<<1, 1>>>(device_blocks, grid_size, device_matrix,
            device_diagonal, device_c, dimension, k, device_scale,
            device_squared_column_length);
        // hipDeviceSynchronize();

        for (int j = k + 1; j < dimension; j++) {
            cuda_parallel_sum_of_products_kernel<<<grid_size, BLOCK_SIZE>>>(
                device_blocks, device_matrix, element_count, starting_index,
                INDEX(k, j, dimension), dimension);
            hipDeviceSynchronize();

            cuda_find_inner_product_and_compute_tau<<<1, 1>>>(device_inner_product, device_blocks, grid_size, 
            device_tau, device_c, k);
            // hipDeviceSynchronize();

            cuda_subtract_tau_product<<<grid_size, BLOCK_SIZE>>>(device_matrix,
                device_tau, device_c, k, j, starting_index, dimension,
                element_count);
            hipDeviceSynchronize();
        }
    }

    bool is_singular;
    hipMemcpy(
        &is_singular, device_is_singular, sizeof(bool), hipMemcpyDeviceToHost);
    cuda_matrix_device_to_host(matrix, device_matrix);
    hipMemcpy(
        diagonal, device_diagonal, diagonal_size, hipMemcpyDeviceToHost);
    hipMemcpy(c, device_c, diagonal_size, hipMemcpyDeviceToHost);

    cuda_matrix_free(device_matrix);
    hipFree(device_diagonal);
    hipFree(device_c);
    hipFree(device_is_singular);
    hipFree(device_inner_product);
    hipFree(device_tau);
    hipFree(device_scale);
    hipFree(device_squared_column_length);
    hipFree(device_blocks);
    return is_singular;
}

#pragma endregion

__global__ void cuda_matrix_qr_decomposition_multi_core_kernel(float *matrix, float *diagonal, float *c, bool *is_singular, int dimension) {
    for (int k = 0; k < dimension; k++)
    {
        // compute scale (parallel reduction: max)
        // singularity check
        // normalize column (independent division)
        // compute column length squared (parallel reduction: sum)
        // do four sequential computations

        for (int j = 0; j < dimension; j++)
        {
            // compute inner product (parallel reduction: sum)
            // tau 
            // subtract tau multiplication from rest of matrix
        }
    }
}

bool cuda_matrix_qr_decomposition_multi_core_single_kernel(
    matrix_t* matrix, float* diagonal, float* c) {
    
    int dimension = matrix->columns;

    device_matrix_t device_matrix =
        cuda_matrix_init(dimension, dimension);
    cuda_matrix_host_to_device(device_matrix, matrix);

    size_t diagonal_size = sizeof(float) * dimension;

    float *device_diagonal;
    hipMalloc(&device_diagonal, diagonal_size);

    float *device_c;
    hipMalloc(&device_c, diagonal_size);

    bool *device_is_singular;
    hipMalloc(&device_is_singular, sizeof(bool));

    int grid_size = (dimension + ELEMENTS_PR_THREAD * BLOCK_SIZE - 1) /
                (ELEMENTS_PR_THREAD * BLOCK_SIZE);

    cuda_matrix_qr_decomposition_multi_core_kernel<<<grid_size, BLOCK_SIZE>>>(
        device_matrix, device_diagonal, device_c, device_is_singular, dimension);

    bool is_singular = false;
    hipMemcpy(
        &is_singular, device_is_singular, sizeof(bool), hipMemcpyDeviceToHost);
    cuda_matrix_device_to_host(matrix, device_matrix);
    hipMemcpy(
        diagonal, device_diagonal, diagonal_size, hipMemcpyDeviceToHost);
    hipMemcpy(c, device_c, diagonal_size, hipMemcpyDeviceToHost);

    cuda_matrix_free(device_matrix);
    hipFree(device_diagonal);
    hipFree(device_c);
    hipFree(device_is_singular);

    return is_singular;
}

bool cuda_matrix_qr_decomposition_multi_core_single_kernel_adapter(algorithm_arg_t *matrix, algorithm_arg_t *diagonal, algorithm_arg_t *c)
{
    return cuda_matrix_qr_decomposition_multi_core_single_kernel(matrix->matrix, diagonal->vector, c->vector);
}