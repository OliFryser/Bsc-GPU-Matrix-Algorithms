extern "C" {
    #include "cuda_matrix_utility.h"
}

#define gpuErrorcheck(function) { gpuAssert((function), __FILE__, __LINE__); }
extern "C" inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Deep copy:
// https://forums.developer.nvidia.com/t/clean-way-of-copying-a-struct-with-pointers-to-the-gpu/225833

// Arguments for why this is bad:
// https://stackoverflow.com/questions/6137218/how-can-i-add-up-two-2d-pitched-arrays-using-nested-for-loops/6137517#6137517

extern "C" device_matrix_t cuda_matrix_init(int rows, int columns) {
    device_matrix_t device_array;
    hipError_t error =
        hipMalloc(&device_array, rows * columns * sizeof(float));
    if (error != hipSuccess) {
        printf("\n%d\n", error);
        return NULL;
    }
    return device_array;
}

extern "C" bool cuda_matrix_free(device_matrix_t device_matrix) {
    if (device_matrix == NULL) return false;
    hipError_t error = hipFree(device_matrix);
    if (error != hipSuccess) {
        printf("\n%d\n", error);
        return false;
    }
    return true;
}

extern "C" bool cuda_matrix_host_to_device(device_matrix_t dst, matrix_t *src) {
    size_t size;
    size = src->rows * src->columns * sizeof(float);
    hipError_t error =
        hipMemcpy(dst, src->values, size, hipMemcpyHostToDevice);

    if (error != hipSuccess) {
        printf("\n%d\n", error);
        return false;
    }

    return true;
}

extern "C" bool cuda_matrix_device_to_host(matrix_t *dst, device_matrix_t src) {
    size_t size;
    size = dst->rows * dst->columns * sizeof(float);
    hipError_t error =
        hipMemcpy(dst->values, src, size, hipMemcpyDeviceToHost);

    if (error != hipSuccess) {
        printf("\n%d\n", error);
        return false;
    }

    return true;
}
