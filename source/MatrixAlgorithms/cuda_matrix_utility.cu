extern "C" {
#include "cuda_matrix_utility.h"
}

extern "C" Matrix *cuda_matrix_init(int rows, int columns) {
    Matrix *host_matrix, *device_matrix;
    float **device_array;
    host_matrix = matrix_init(rows, columns);

    hipMalloc(&device_matrix, sizeof(Matrix));

    hipMemcpy(device_matrix, host_matrix, sizeof(Matrix),
               hipMemcpyHostToDevice);

    hipMalloc(&device_array, rows * sizeof(float *));

    for (int i = 0; i < rows; i++) {
        float *row;
        hipMalloc(&row, columns * sizeof(float));
        hipMalloc(&(device_array[i]), columns * sizeof(float));
    }

    // hipMemcpy(device_matrix->values, device_array, rows * sizeof(float *),
    // hipMemcpyDeviceToDevice);
    return device_matrix;
}

extern "C" void cuda_matrix_free(Matrix *device_matrix) {
    if (device_matrix == NULL) return;
    if (device_matrix->values != NULL) {
        for (int i = 0; i < device_matrix->rows; i++) {
            if (device_matrix->values[i] != NULL)
                hipFree(device_matrix->values[i]);
        }
        hipFree(device_matrix->values);
    }
    hipFree(device_matrix);
}

extern "C" void cuda_matrix_host_to_device(Matrix *dst, Matrix *src) {}
extern "C" void cuda_matrix_device_to_host(Matrix *dst, Matrix *src) {}