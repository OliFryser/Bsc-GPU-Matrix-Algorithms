extern "C" {
#include "cuda_matrix_utility.h"
}

// Deep copy:
// https://forums.developer.nvidia.com/t/clean-way-of-copying-a-struct-with-pointers-to-the-gpu/225833

// Arguments for why this is bad:
// https://stackoverflow.com/questions/6137218/how-can-i-add-up-two-2d-pitched-arrays-using-nested-for-loops/6137517#6137517
extern "C" DEVICE_MATRIX cuda_matrix_init(int rows, int columns) {
    DEVICE_MATRIX device_array;
    hipMalloc(&device_array, rows * columns * sizeof(float));
    return device_array;
}

extern "C" void cuda_matrix_free(DEVICE_MATRIX device_matrix) {
    if (device_matrix == NULL) return;
    hipFree(device_matrix);
}

void cuda_matrix_2d_to_1d(float *dst, Matrix *src) {
    for (int i = 0; i < src->rows; i++)
        for (int j = 0; j < src->columns; j++)
            dst[INDEX(i, j, src->columns)] = src->values[i][j];
}

void cuda_matrix_1d_to_2d(Matrix *dst, float *src) {
    for (int i = 0; i < dst->rows; i++)
        for (int j = 0; j < dst->columns; j++)
            dst->values[i][j] = src[INDEX(i, j, dst->columns)];
}

extern "C" void cuda_matrix_host_to_device(DEVICE_MATRIX dst, Matrix *src) {
    float *cpu_values;
    size_t size = src->rows * src->columns * sizeof(float);
    cpu_values = (float *)malloc(size);
    cuda_matrix_2d_to_1d(cpu_values, src);
    hipMemcpy(dst, cpu_values, size, hipMemcpyHostToDevice);
    free(cpu_values);
}

extern "C" void cuda_matrix_device_to_host(Matrix *dst, DEVICE_MATRIX src) {
    float *cpu_values;
    size_t size = dst->rows * dst->columns * sizeof(float);
    cpu_values = (float *)malloc(size);
    hipMemcpy(cpu_values, src, size, hipMemcpyDeviceToHost);
    cuda_matrix_1d_to_2d(dst, cpu_values);
    free(cpu_values);
}
