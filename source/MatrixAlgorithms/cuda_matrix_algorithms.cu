#include "hip/hip_runtime.h"
extern "C" {
#include "cuda_matrix_algorithms.h"
}

__global__ void cuda_matrix_addition_single_core_kernel(
    device_matrix_t matrix_a, device_matrix_t matrix_b,
    device_matrix_t matrix_c, int size, int rows, int columns) {
    for (int i = 0; i < size; i++) {
        matrix_c[i] = matrix_a[i] + matrix_b[i];
    }
}

__global__ void cuda_matrix_addition_multi_core_kernel(device_matrix_t matrix_a,
    device_matrix_t matrix_b, device_matrix_t matrix_c, int size, int rows,
    int columns) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    matrix_c[index] = matrix_a[index] + matrix_b[index];
}

__global__ void cuda_matrix_addition_multi_core_kernel2(
    device_matrix_t matrix_a, device_matrix_t matrix_b,
    device_matrix_t matrix_c, int size, int rows, int columns) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= rows || j >= columns) return;

    matrix_c[INDEX(i, j, columns)] =
        matrix_a[INDEX(i, j, columns)] + matrix_b[INDEX(i, j, columns)];
}

__global__ void cuda_matrix_multiplication_single_core_kernel(
    device_matrix_t matrix_a, device_matrix_t matrix_b,
    device_matrix_t matrix_c, int l, int n, int m) {
    float sum_of_products;

    for (int i = 0; i < l; i++)
        for (int j = 0; j < n; j++) {
            sum_of_products = 0.0f;
            for (int k = 0; k < m; k++)
                sum_of_products +=
                    matrix_a[INDEX(i, k, m)] * matrix_b[INDEX(k, j, n)];
            matrix_c[INDEX(i, j, n)] = sum_of_products;
        }
}

__global__ void cuda_matrix_multiplication_multicore_unwrapping_i_kernel(
    device_matrix_t matrix_a, device_matrix_t matrix_b,
    device_matrix_t matrix_c, int l, int n, int m) {
    int i = blockIdx.x;
    float sum_of_products;
    
    for (int j = 0; j < n; j++) {
        sum_of_products = 0.0f;
        for (int k = 0; k < m; k++)
            sum_of_products +=
                matrix_a[INDEX(i, k, m)] * matrix_b[INDEX(k, j, n)];
        matrix_c[INDEX(i, j, n)] = sum_of_products;
    }
}

__global__ void cuda_matrix_multiplication_multicore_unwrapping_i_and_j_kernel(
    device_matrix_t matrix_a, device_matrix_t matrix_b,
    device_matrix_t matrix_c, int l, int n, int m) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    float sum_of_products = 0.0f;

    for (int k = 0; k < m; k++)
        sum_of_products += matrix_a[INDEX(i, k, m)] * matrix_b[INDEX(k, j, n)];

    matrix_c[INDEX(i, j, n)] = sum_of_products;
}

#define BLOCK_SIZE 16

__device__ device_matrix_t get_sub_matrix(
    device_matrix_t matrix, int row, int column, int width) {
    return &matrix[INDEX(row * BLOCK_SIZE, column * BLOCK_SIZE, width)];
}

__device__ void print_device_matrix(
    device_matrix_t matrix, int row, int column, int width) {
    printf("\n\n");
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < column; j++) {
            printf("%f ", matrix[INDEX(i, j, width)]);
        }
        printf("\n");
    }
    printf("Done");
}

__device__ void print_shared_matrix(float shared_matrix[BLOCK_SIZE][BLOCK_SIZE]) {
    printf("\n");
    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        for (int j = 0; j < BLOCK_SIZE; j++)
        {
            printf("%f ", shared_matrix[j][i]);
        }
        printf("\n");
    }
}

__global__ void cuda_matrix_multiplication_multi_core_shared_memory_kernel(
    device_matrix_t matrix_a, device_matrix_t matrix_b,
    device_matrix_t matrix_c, int l, int n, int m) {
    
    int block_row = blockIdx.y;
    int block_column = blockIdx.x;
    int row = threadIdx.y;
    int column = threadIdx.x;
    float c_value = .0f;

    int subs_in_m = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (int k = 0; k < subs_in_m; k++) {
        device_matrix_t a_sub = get_sub_matrix(matrix_a, block_row, k, m);
        __shared__ float shared_a_sub[BLOCK_SIZE][BLOCK_SIZE];
        shared_a_sub[row][column] = a_sub[INDEX(row, column, m)];

        device_matrix_t b_sub = get_sub_matrix(matrix_b, k, block_column, n);
        __shared__ float shared_b_sub[BLOCK_SIZE][BLOCK_SIZE];
        shared_b_sub[row][column] = b_sub[INDEX(row, column, n)];
        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; i++) 
            c_value += shared_a_sub[row][i] * shared_b_sub[i][column];
        __syncthreads();
    }

    if (row + BLOCK_SIZE * block_row < l && column + BLOCK_SIZE * block_column  < n) {
        device_matrix_t c_sub = get_sub_matrix(matrix_c, block_row, block_column, n);
        c_sub[INDEX(row, column, n)] = c_value;
    } 
}

__global__ void cuda_matrix_multiplication_multi_core_shared_memory_fewer_accesses_kernel(
    device_matrix_t matrix_a, device_matrix_t matrix_b, device_matrix_t matrix_c, int l, int n, int m) {
    
    int block_row = blockIdx.y;
    int block_column = blockIdx.x;
    int row = threadIdx.y;
    int column = threadIdx.x;
    float c_value = .0f;

    // Find the top left corner of the sub matrix
    // Then find the row inside the sub matrix
    // Then find the column inside the sub matrix
    device_matrix_t a_sub = &matrix_a[block_row * BLOCK_SIZE * m + row * m + column];
    device_matrix_t b_sub = &matrix_b[block_column * BLOCK_SIZE + row * n + column];

    int subs_in_m = m + BLOCK_SIZE - 1;
    for (int k = 0; k < subs_in_m; k += BLOCK_SIZE) {
        __shared__ float shared_a_sub[BLOCK_SIZE][BLOCK_SIZE];
        shared_a_sub[row][column] = a_sub[k];

        __shared__ float shared_b_sub[BLOCK_SIZE][BLOCK_SIZE];
        shared_b_sub[row][column] = b_sub[k * n];
        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; i++) 
            c_value += shared_a_sub[row][i] * shared_b_sub[i][column];
        __syncthreads();
    }

    if (row + BLOCK_SIZE * block_row < l && column + BLOCK_SIZE * block_column  < n) {
        device_matrix_t c_sub = get_sub_matrix(matrix_c, block_row, block_column, n);
        c_sub[INDEX(row, column, n)] = c_value;
    } 
}

__global__ void cuda_matrix_qr_decomposition_single_core_kernel(device_matrix_t matrix, float *diagonal, float *c, 
    int dimension, bool *is_singular) {

    float column_length;  // sigma in book
    float column_length_squared, element;
    int n = dimension;
    float scale;
    *is_singular = false;

    // for every column
    for (int k = 0; k < n; k++) {
        scale = 0.0f;
        // scale is the max absolute value of the column
        for (int i = k; i < n; i++)
            scale = fmaxf(scale, fabsf(matrix[INDEX(i, k, n)]));

        if (scale == 0.0) {
            *is_singular = true;
            c[k] = diagonal[k] = 0.0f;
            continue;
        }
        // Normalize column
        for (int i = k; i < n; i++) matrix[INDEX(i, k, n)] /= scale;

        // column length below diagonal
        column_length_squared = 0.0f;  // sum in book.
        for (int i = k; i < n; i++) {
            element = matrix[INDEX(i, k, n)];
            column_length_squared += element * element;
        }

        // column length below diagonal, with the sign of diagonal k
        column_length =
            SIGN(sqrtf(column_length_squared), matrix[INDEX(k, k, n)]);

        // add column length to diagonal k
        matrix[INDEX(k, k, n)] += column_length;

        c[k] = matrix[INDEX(k, k, n)] * column_length;

        diagonal[k] = -scale * column_length;

        // Calculate Q[k] = I - (u[k] (x) u[k]) / c[k]
        for (int j = k + 1; j < n; j++) {
            // inner product for column j below diagonal
            float inner_product = 0.0f;
            for (int i = k; i < n; i++) {
                inner_product += matrix[(INDEX(i, k, n))] *
                                 matrix[(INDEX(i, j, n))];
            }

            // division
            float tau = inner_product / c[k];

            for (int i = k; i < n; i++) {
                matrix[(INDEX(i, j, n))] -=
                    tau * matrix[(INDEX(i, k, n))];
            }
        }
    }

    if (!*is_singular) *is_singular = diagonal[n - 1] == 0.0f;
}

bool cuda_qr_decomposition_runner(matrix_t *matrix, float *diagonal, float *c, 
    void (*kernel)(device_matrix_t, float *, float *, int, bool *), dim3 grid_size, dim3 block_size) {

    device_matrix_t device_matrix = cuda_matrix_init(matrix->rows, matrix->columns);
    cuda_matrix_host_to_device(device_matrix, matrix);

    size_t diagonal_size = sizeof(float) * matrix->columns;
    
    float *device_diagonal;
    hipMalloc(&device_diagonal, diagonal_size);

    float *device_c;
    hipMalloc(&device_c, diagonal_size);

    bool *device_is_singular;
    hipMalloc(&device_is_singular, sizeof(bool));

    kernel<<<grid_size, block_size>>>(device_matrix, device_diagonal, device_c, matrix->columns, device_is_singular);

    bool is_singular;
    hipMemcpy(&is_singular, device_is_singular, sizeof(bool), hipMemcpyDeviceToHost);
    cuda_matrix_device_to_host(matrix, device_matrix);
    hipMemcpy(diagonal, device_diagonal, diagonal_size, hipMemcpyDeviceToHost);
    hipMemcpy(c, device_c, diagonal_size, hipMemcpyDeviceToHost);

    cuda_matrix_free(device_matrix);
    hipFree(device_diagonal);
    hipFree(device_c);
    hipFree(device_is_singular);

    return is_singular;
}

bool cuda_matrix_algorithm_runner(matrix_t* matrix_a, matrix_t* matrix_b,
    matrix_t* matrix_c, int kernel_param1, int kernel_param2, int kernel_param3,
    void (*kernel)(
        device_matrix_t, device_matrix_t, device_matrix_t, int, int, int),
    dim3 grid_size, dim3 block_size) {
    if (matrix_a == NULL || matrix_b == NULL || matrix_c == NULL) return false;

    device_matrix_t device_matrix_a =
        cuda_matrix_init(matrix_a->rows, matrix_a->columns);
    device_matrix_t device_matrix_b =
        cuda_matrix_init(matrix_b->rows, matrix_b->columns);
    device_matrix_t device_matrix_c =
        cuda_matrix_init(matrix_c->rows, matrix_c->columns);

    if (device_matrix_a == NULL || device_matrix_b == NULL ||
        device_matrix_c == NULL)
        return false;

    cuda_matrix_host_to_device(device_matrix_a, matrix_a);
    cuda_matrix_host_to_device(device_matrix_b, matrix_b);

    kernel<<<grid_size, block_size>>>(device_matrix_a, device_matrix_b,
        device_matrix_c, kernel_param1, kernel_param2, kernel_param3);

    cuda_matrix_device_to_host(matrix_c, device_matrix_c);

    cuda_matrix_free(device_matrix_a);
    cuda_matrix_free(device_matrix_b);
    cuda_matrix_free(device_matrix_c);

    return true;
}

bool cuda_matrix_addition_single_core_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_addition_single_core(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_addition_single_core(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    return cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_c->rows * matrix_c->columns, matrix_c->rows, matrix_c->columns,
        &(cuda_matrix_addition_single_core_kernel), dim3(1), dim3(1));
}

bool cuda_matrix_addition_multi_core_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_addition_multi_core(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_addition_multi_core(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    bool success;
    dim3 grid_size, block_size;
    grid_size = dim3(matrix_a->rows);
    block_size = dim3(matrix_a->columns);

    success = cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_c->rows * matrix_c->columns, matrix_c->rows, matrix_c->columns,
        &(cuda_matrix_addition_multi_core_kernel), grid_size, block_size);

    return success;
}

bool cuda_matrix_addition_multi_core2_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_addition_multi_core2(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_addition_multi_core2(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    bool success;
    dim3 grid_size, block_size;
    int threads_per_block_dim = 16;

    block_size = dim3(threads_per_block_dim, threads_per_block_dim);
    grid_size = dim3((matrix_a->rows + block_size.x - 1) / block_size.x,
        (matrix_a->columns + block_size.y - 1) / block_size.y);

    success = cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_c->rows * matrix_c->columns, matrix_c->rows, matrix_c->columns,
        &(cuda_matrix_addition_multi_core_kernel2), grid_size, block_size);

    return success;
}

bool cuda_matrix_multiplication_single_core_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_multiplication_single_core(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_multiplication_single_core(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    return cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_a->rows, matrix_b->columns, matrix_a->columns,
        &cuda_matrix_multiplication_single_core_kernel, dim3(1), dim3(1));
}

bool cuda_matrix_multiplication_multi_core_unwrapping_i_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_multiplication_multi_core_unwrapping_i(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_multiplication_multi_core_unwrapping_i(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    return cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_a->rows, matrix_b->columns, matrix_a->columns,
        &cuda_matrix_multiplication_multicore_unwrapping_i_kernel,
        dim3(matrix_a->rows), dim3(1));
}

bool cuda_matrix_multiplication_multi_core_unwrapping_i_and_j_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_multiplication_multi_core_unwrapping_i_and_j(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_multiplication_multi_core_unwrapping_i_and_j(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    return cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_a->rows, matrix_b->columns, matrix_a->columns,
        &cuda_matrix_multiplication_multicore_unwrapping_i_and_j_kernel,
        dim3(matrix_a->rows), dim3(matrix_b->columns));
}

bool cuda_matrix_multiplication_multi_core_shared_memory_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_multiplication_multi_core_shared_memory(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_multiplication_multi_core_shared_memory_fewer_accesses_adapter(algorithm_arg_t *arg_a, algorithm_arg_t *arg_b, algorithm_arg_t *arg_c) {
    return cuda_matrix_multiplication_multi_core_shared_memory_fewer_accesses(arg_a->matrix, arg_b->matrix, arg_c->matrix);
}

bool cuda_matrix_multiplication_multi_core_shared_memory(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    bool success;
    dim3 block_dim, grid_dim;

    block_dim = dim3(BLOCK_SIZE, BLOCK_SIZE);

    grid_dim = dim3((matrix_b->columns + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (matrix_a->rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    success = cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_a->rows, matrix_b->columns, matrix_a->columns,
        &(cuda_matrix_multiplication_multi_core_shared_memory_kernel), grid_dim,
        block_dim);

    return success;
}

bool cuda_matrix_multiplication_multi_core_shared_memory_fewer_accesses(
    matrix_t* matrix_a, matrix_t* matrix_b, matrix_t* matrix_c) {
    bool success;
    dim3 block_dim, grid_dim;

    block_dim = dim3(BLOCK_SIZE, BLOCK_SIZE);

    grid_dim = dim3((matrix_b->columns + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (matrix_a->rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    success = cuda_matrix_algorithm_runner(matrix_a, matrix_b, matrix_c,
        matrix_a->rows, matrix_b->columns, matrix_a->columns,
        &(cuda_matrix_multiplication_multi_core_shared_memory_fewer_accesses_kernel), grid_dim,
        block_dim);

    return success;
}

bool cuda_matrix_qr_decomposition_single_core(matrix_t *matrix, float *diagonal, float *c)
{
    return cuda_qr_decomposition_runner(matrix, diagonal, c, 
    &(cuda_matrix_qr_decomposition_single_core_kernel), dim3(1), dim3(1));
}
