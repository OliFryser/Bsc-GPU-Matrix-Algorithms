#include "hip/hip_runtime.h"
extern "C" {
#include "cuda_matrix_algorithms.h"
}

__global__ void matrix_addition_gpu_single_core_kernel(DEVICE_MATRIX matrix1,
    DEVICE_MATRIX matrix2, DEVICE_MATRIX result, int size) {
    for (int i = 0; i < size; i++) {
        result[i] = matrix1[i] + matrix2[i];
    }
}

__global__ void matrix_addition_gpu_multi_core_kernel(DEVICE_MATRIX matrix1,
    DEVICE_MATRIX matrix2, DEVICE_MATRIX result, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    result[index] = matrix1[index] + matrix2[index];
}

bool gpu_matrix_algorithm_runner(Matrix* matrix1, Matrix* matrix2,
    Matrix* result,
    void (*kernel)(DEVICE_MATRIX, DEVICE_MATRIX, DEVICE_MATRIX, int),
    int grid_size, int block_size) {
    if (matrix1 == NULL || matrix2 == NULL || result == NULL) return false;

    DEVICE_MATRIX device_matrix1 =
        cuda_matrix_init(matrix1->rows, matrix1->columns);
    DEVICE_MATRIX device_matrix2 =
        cuda_matrix_init(matrix2->rows, matrix2->columns);
    DEVICE_MATRIX device_result =
        cuda_matrix_init(result->rows, result->columns);

    if (device_matrix1 == NULL || device_matrix2 == NULL ||
        device_result == NULL)
        return false;

    cuda_matrix_host_to_device(device_matrix1, matrix1);
    cuda_matrix_host_to_device(device_matrix2, matrix2);
    cuda_matrix_host_to_device(device_result, result);

    kernel<<<grid_size, block_size>>>(device_matrix1, device_matrix2,
        device_result, result->rows * result->columns);

    cuda_matrix_device_to_host(result, device_result);

    cuda_matrix_free(device_matrix1);
    cuda_matrix_free(device_matrix2);
    cuda_matrix_free(device_result);

    return true;
}

extern "C" bool matrix_addition_gpu_single_core(
    Matrix* matrix1, Matrix* matrix2, Matrix* result) {
    return gpu_matrix_algorithm_runner(matrix1, matrix2, result,
        &(matrix_addition_gpu_single_core_kernel), 1, 1);
}

bool matrix_addition_gpu_multi_core(
    Matrix* matrix1, Matrix* matrix2, Matrix* result) {
    bool success;
    int grid_size, block_size;
    grid_size = matrix1->rows;
    block_size = matrix1->columns;

    success = gpu_matrix_algorithm_runner(matrix1, matrix2, result,
        &(matrix_addition_gpu_multi_core_kernel), grid_size, block_size);

    return success;
}